#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "opencv2/core/core.hpp"  
#include "opencv2/highgui/highgui.hpp"  
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
using namespace cv;
using namespace std;

__global__ void convolve(unsigned char* imgInput, float* mask, int width, int height, int paddingX, int paddingY, int kWidth, int kHeight, unsigned char* imgOutput)
{
	// Calculate our pixel's location
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	float sum = 0.0;
	int   pWidth = kWidth / 2;
	int   pHeight = kHeight / 2;

	//Solo ejecuta validos pixeles
	if (x >= pWidth + paddingX && y >= pHeight + paddingY && x < (blockDim.x * gridDim.x) - pWidth - paddingX &&
		y < (blockDim.y * gridDim.y) - pHeight - paddingY)
	{
		for (int j = -pHeight; j <= pHeight; j++)
		{
			for (int i = -pWidth; i <= pWidth; i++)
			{
				// Sample the weight for this location
				int ki = (i + pWidth);
				int kj = (j + pHeight);
				float w = mask[(kj * kWidth) + ki];


				sum += w * float(imgInput[((y + j) * width) + (x + i)]);
			}
		}
	}
	imgOutput[(y * width) + x] = (unsigned char)sum;
}

__global__ void pythagoras(unsigned char* Gx, unsigned char* Gy, unsigned char* G, unsigned char* theta)
{
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	float af = float(Gx[idx]);
	float bf = float(Gy[idx]);

	G[idx] = (unsigned char)sqrtf(af * af + bf * bf);
	theta[idx] = (unsigned char)atan2f(af, bf)*63.994;

}
__global__ void greyConvertor(uchar3* const imgInput, unsigned char* const imgOutput, int imgheight, int imgwidth) {
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

	if (idx < imgwidth && idy < imgheight)
	{
		uchar3 rgb_Val = imgInput[idy * imgwidth + idx];
		imgOutput[idy * imgwidth + idx] = 0.299f * rgb_Val.x + 0.587f * rgb_Val.y + 0.114f * rgb_Val.z;
	}
}

__host__ unsigned char* createImageBuffer(unsigned int bytes, unsigned char** devicePtr)
{
	unsigned char* ptr = NULL;
	hipSetDeviceFlags(hipDeviceMapHost);
	hipHostAlloc(&ptr, bytes, hipHostMallocMapped);
	hipHostGetDevicePointer(devicePtr, ptr, 0);
	return ptr;
}

__host__ int compute() {
	hipDeviceProp_t propertise;
	hipGetDeviceProperties(&propertise, 0);
	int blocksPerSM = propertise.maxThreadsPerMultiProcessor / propertise.maxThreadsPerBlock;
	int threadsPerSM = propertise.maxThreadsPerMultiProcessor;
	int dimension = 1;
	float result = 0;
	while (result <= 1.0) {
		dimension *= 2;
		result = (blocksPerSM*dimension*dimension) / threadsPerSM;
	}
	return dimension / 2;
}


int main() {

	Mat inputImage = imread("remastered-lena-512x512.tiff");
	if (inputImage.empty())
	{
		printf("!!! Failed imread(): image not found\n");
		exit(1);
	}
	imshow("originalimage", inputImage);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float* d_gaussianFilter = nullptr;
	float* d_sobelFilter_Gx = nullptr;
	float* d_sobelFilter_Gy = nullptr;


	const float gaussianFilter[25] = {
		2.f / 159.f,  4.f / 159.f,  5.f / 159.f,  4.f / 159.f, 2.f / 159.f,
		4.f / 159.f,  9.f / 159.f, 12.f / 159.f,  9.f / 159.f, 4.f / 159.f,
		5.f / 159.f, 12.f / 159.f, 15.f / 159.f, 12.f / 159.f, 5.f / 159.f,
		4.f / 159.f,  9.f / 159.f, 12.f / 159.f,  9.f / 159.f, 4.f / 159.f,
		2.f / 159.f,  4.f / 159.f,  5.f / 159.f,  4.f / 159.f, 2.f / 159.f,
	};

	const float sobleFilter_Gx[9] = {
		-1.f, 0.f, 1.f,
		-2.f, 0.f, 2.f,
		-1.f, 0.f, 1.f,
	};

	const float sobelFilter_Gy[9] = {
		1.f, 2.f, 1.f,
		0.f, 0.f, 0.f,
		-1.f, -2.f, -1.f,
	};

	int imgHeight = inputImage.rows;
	int imgWidth = inputImage.cols;
	int imgChannels = inputImage.channels();

	hipEventRecord(start);//start time-----------------------

	hipMalloc(&d_gaussianFilter, sizeof(gaussianFilter));
	hipMalloc(&d_sobelFilter_Gx, sizeof(sobleFilter_Gx));
	hipMalloc(&d_sobelFilter_Gy, sizeof(sobelFilter_Gy));

	hipMemcpy(d_gaussianFilter, gaussianFilter, 25 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_sobelFilter_Gx, sobleFilter_Gx, 9 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_sobelFilter_Gy, sobelFilter_Gy, 9 * sizeof(float), hipMemcpyHostToDevice);

	unsigned char* sourceData = nullptr, *blurredData = nullptr, *sobel_G_Data = nullptr, *sobel_theta_Data = nullptr;
	Mat source(inputImage.size(), CV_8U, createImageBuffer(imgWidth * imgHeight, &sourceData));
	Mat blurred(inputImage.size(), CV_8U, createImageBuffer(imgWidth * imgHeight, &blurredData));
	Mat sobel_G(inputImage.size(), CV_8U, createImageBuffer(imgWidth * imgHeight, &sobel_G_Data));
	Mat sobel_theta(inputImage.size(), CV_8U, createImageBuffer(imgWidth * imgHeight, &sobel_theta_Data));


	uchar3* d_input = nullptr;
	hipMalloc((void**)&d_input, imgWidth * imgHeight*sizeof(uchar3));
	hipMemcpy(d_input, inputImage.data, imgWidth * imgHeight * sizeof(uchar3), hipMemcpyHostToDevice);

	unsigned char* d_Gx = nullptr;
	unsigned char* d_Gy = nullptr;
	hipMalloc(&d_Gx, imgWidth * imgHeight);
	hipMalloc(&d_Gy, imgWidth * imgHeight);

	// compute optimeze kernel configuration
	int optimalFactor = compute();
	printf("optimal factor for your gpu is: %d \n", optimalFactor);
	dim3 DimBlock(optimalFactor, optimalFactor);
	dim3 DimGrid((imgWidth + DimBlock.x - 1) / DimBlock.x, (imgHeight + DimBlock.y - 1) / DimBlock.y);
	//--------------------------------------

	dim3 pBlocks(inputImage.size().width * inputImage.size().height / 256);
	dim3 pThreads(256, 1);

	greyConvertor << <DimGrid, DimBlock >> > (d_input, sourceData, imgWidth, imgHeight);
	convolve << <DimGrid, DimBlock >> > (sourceData, d_gaussianFilter, imgWidth, imgHeight, 0, 0, 5, 5, blurredData);

	// sobel gradient convolutions (x&y padding is now 2 because there is a border of 2 around a 5x5 gaussian filtered image)
	convolve << <DimGrid, DimBlock >> > (blurredData, d_sobelFilter_Gx, imgWidth, imgHeight, 2, 2, 3, 3, d_Gx);
	convolve << <DimGrid, DimBlock >> > (blurredData, d_sobelFilter_Gy, imgWidth, imgHeight, 2, 2, 3, 3, d_Gy);
	pythagoras << <pBlocks, pThreads >> > (d_Gx, d_Gy, sobel_G_Data, sobel_theta_Data);
	hipDeviceSynchronize();

	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float miliSeconds;
	hipEventElapsedTime(&miliSeconds, start, stop);
	printf("kernel execution time: %f ms", miliSeconds);
	imshow("GreyImage", source);
	imshow("gaussianFilterImage", blurred);
	imshow("sobelFilter_G", sobel_G);
	imshow("sobelFilter-theta", sobel_theta);
	waitKey(0);

	//cleanup
	hipHostFree(source.data);
	hipHostFree(blurred.data);
	hipHostFree(sobel_G.data);
	hipHostFree(sobel_theta.data);
	hipFree(d_gaussianFilter);
	hipFree(d_sobelFilter_Gx);
	hipFree(d_sobelFilter_Gy);
	hipFree(d_input);
	hipFree(d_Gx);
	hipFree(d_Gy);

	return 0;

}
